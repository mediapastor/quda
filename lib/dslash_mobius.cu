#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>

#include <inline_ptx.h>

#include <hipblas.h>
#include <mma.h>

namespace quda {

  namespace mobius {

    template<class T>
    struct MDWFSharedMemory
    {
      __device__ inline operator T*()
      {
        extern __shared__ int __smem[];
        return (T*)__smem;
      }

      __device__ inline operator const T*() const
      {
        extern __shared__ int __smem[];
        return (T*)__smem;
      }
    };

#undef GPU_STAGGERED_DIRAC
#include <dslash_constants.h>
#include <dslash_textures.h>
#include <dslash_index.cuh>

    // Enable shared memory dslash for Fermi architecture
    //#define SHARED_WILSON_DSLASH
    //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#ifdef GPU_DOMAIN_WALL_DIRAC
#include <mdw_dslash4_def.h>      // Dslash4, intermediate operator for Mobius Mat_4 kernels
#include <mdw_dslash4pre_def.h>   // Dslash4pre, intermediate operator for Mobius Mat_4 kernels
#include <mdw_dslash5_def.h>      // Dslash5 Mobius Domain Wall kernels
#include <mdw_dslash5inv_def.h>   // Dslash5inv Mobius Domain Wall kernels
#include <mdw_dslash4_dslash5inv_dslash4pre_def.h>   // Dslash5inv Mobius Domain Wall kernels
#include <mdw_dslash4_dslash5inv_xpay_dslash5inv_dagger_def.h>   // Dslash5inv Mobius Domain Wall kernels
#include <mdw_dslash4_dagger_dslash4pre_dagger_dslash5inv_dagger_def.h>
#include <mdw_dslash4_dagger_dslash4pre_dagger_xpay_def.h>
#include <mdw_dslash5inv_def_sm.h>
#include <mdw_dslash5inv_def_sm_tc.h>
#endif

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#include <dslash_quda.cuh>
  }

  // declare the dslash events
#include <dslash_events.cuh>

  using namespace mobius;

#ifdef GPU_DOMAIN_WALL_DIRAC
  //Dslash class definition for Mobius Domain Wall Fermion
  template <typename sFloat, typename gFloat>
  class MDWFDslashPCCuda : public DslashCuda {

  private:
    const int DS_type;

    bool checkGrid(TuneParam &param) const {
      if (param.grid.x > (unsigned int)deviceProp.maxGridSize[0] || param.grid.y > (unsigned int)deviceProp.maxGridSize[1]) {
        warningQuda("Autotuner is skipping blockDim=(%u,%u,%u), gridDim=(%u,%u,%u) because lattice volume is too large",
		    param.block.x, param.block.y, param.block.z, 
		    param.grid.x, param.grid.y, param.grid.z);
        return false;
      } else {
        return true;
      }
    }

  protected:
    bool advanceBlockDim(TuneParam &param) const
    {
//      const unsigned int max_shared = deviceProp.sharedMemPerBlock;
      const unsigned int max_shared = deviceProp.major>=7 ? 96*1024 : deviceProp.sharedMemPerBlock;
//      const int step[2] = { deviceProp.warpSize, 1 };
      const int step[2] = { 16, 1 };
      bool advance[2] = { false, false };

      // first try to advance block.x
      param.block.x += step[0];
      if (param.block.x > (unsigned int)deviceProp.maxThreadsDim[0] ||
          shared_bytes_per_block(param.block.x, param.block.y) > max_shared) {
        advance[0] = false;
        param.block.x = step[0]; // reset block.x
      } else {
        advance[0] = true; // successfully advanced block.x
      }
      
      if(DS_type < 4){
        if (!advance[0]) {  // if failed to advance block.x, now try block.y
          param.block.y += step[1];
  
          if (param.block.y > (unsigned)in->X(4) ||
              sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
            advance[1] = false;
            param.block.y = step[1]; // reset block.x
          } else {
            advance[1] = true; // successfully advanced block.y
          }
        }
      }

      if (advance[0] || advance[1]) {
        param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			   (in->X(4)+param.block.y-1) / param.block.y, 1);

        param.shared_bytes = shared_bytes_per_block(param.block.x, param.block.y);
        
        bool advance = true;
        if (!checkGrid(param)) advance = advanceBlockDim(param);
        return advance;
      } else {
        return false;
      }
    }

		bool advanceSharedBytes(TuneParam &param) const
    {
      if (tuneSharedBytes()) {
//				const int max_shared = deviceProp.sharedMemPerBlock;
				const unsigned int max_shared = deviceProp.major>=7 ? 96*1024 : deviceProp.sharedMemPerBlock;
				const int max_blocks_per_sm = std::min(deviceProp.maxThreadsPerMultiProcessor / (param.block.x*param.block.y*param.block.z), maxBlocksPerSM());
				int blocks_per_sm = max_shared / (param.shared_bytes ? param.shared_bytes : 1);
				if (blocks_per_sm > max_blocks_per_sm) blocks_per_sm = max_blocks_per_sm;
				param.shared_bytes = (blocks_per_sm > 0 ? max_shared / blocks_per_sm + 1 : max_shared + 1);
			
				if ((size_t)param.shared_bytes > max_shared) {
				  TuneParam next(param);
				  advanceBlockDim(next); // to get next blockDim
				  int nthreads = next.block.x * next.block.y * next.block.z;
				  param.shared_bytes = shared_bytes_per_block(next.block.x,next.block.y) > sharedBytesPerBlock(param) ?
				     shared_bytes_per_block(next.block.x,next.block.y) : sharedBytesPerBlock(param);
				  return false;
				} else {
				  return true;
				}
      } else {
				return false;
      }
    }

    unsigned int sharedBytesPerThread() const { 
      if(DS_type >= 4){
        return 24*(in->Precision()==8?8:4);
      }else{
        return 0;
      }
    }
 
    unsigned int shared_bytes_per_block(int x, int y) const { 
      if(DS_type == 9){
        return ( (y*4)*(y*4+8)+(y*4)*(x*6+8)*2 )*2; // 4*4*2 TODO: fix this!
      }else{
        return sharedBytesPerThread()*x*y;
      }
    }
 
  public:
    MDWFDslashPCCuda(cudaColorSpinorField *out, const GaugeField &gauge, const cudaColorSpinorField *in,
		     const cudaColorSpinorField *x, const double mferm, const double a,
                     const double *b_5, const double *c_5, const double m5,
                     const int parity, const int dagger, const int *commOverride, const int DS_type)
      : DslashCuda(out, in, x, gauge, parity, dagger, commOverride), DS_type(DS_type)
    { 
      dslashParam.a = a;
      dslashParam.a_f = a;
      dslashParam.mferm = mferm;
      dslashParam.mferm_f = mferm;

      memcpy(dslashParam.mdwf_b5_d, b_5, out->X(4)*sizeof(double));
      memcpy(dslashParam.mdwf_c5_d, c_5, out->X(4)*sizeof(double));
      for (int s=0; s<out->X(4); s++) {
        dslashParam.mdwf_b5_f[s] = (float)dslashParam.mdwf_b5_d[s];
        dslashParam.mdwf_c5_f[s] = (float)dslashParam.mdwf_c5_d[s];
      }

      dslashParam.m5_d = m5;
      dslashParam.m5_f = (float)m5;
    }
    virtual ~MDWFDslashPCCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      if(dslashParam.partial_length){
        
        char config[256];
        switch(DS_type){
          case 0:
            if(dslashParam.expanding){
              sprintf(config, ",Dslash4,partial%d,%d,%d,%d,expand%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3],
                dslashParam.Rz[0], dslashParam.Rz[1], dslashParam.Rz[2], dslashParam.Rz[3]);
            }else{
              sprintf(config, ",Dslash4,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            }
            strcat(key.aux,config);
            break;
          case 1:
            sprintf(config, ",Dslash4pre,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            strcat(key.aux,config);
            break;
          case 2:
            sprintf(config, ",Dslash5,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            strcat(key.aux,config);
            break;
          case 3:
            sprintf(config, ",Dslash5inv,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            strcat(key.aux,config);
          	break;
					case 4:
            if(dslashParam.expanding){
              sprintf(config, ",Dslash4Dslash5invDslash4pre,partial%d,%d,%d,%d,expand%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3],
                dslashParam.Rz[0], dslashParam.Rz[1], dslashParam.Rz[2], dslashParam.Rz[3]);
            }else{
              sprintf(config, ",Dslash4Dslash5invDslash4pre,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            }
            strcat(key.aux,config);
            break;
					case 5:
            if(dslashParam.expanding){
              sprintf(config, ",Dslash4Dslash5invXpayDslash5invDagger,partial%d,%d,%d,%d,expand%d,%d,%d,%d", 
								dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3],
                dslashParam.Rz[0], dslashParam.Rz[1], dslashParam.Rz[2], dslashParam.Rz[3]);
            }else{
              sprintf(config, ",Dslash4Dslash5invXpayDslash5invDagger,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            }
            strcat(key.aux,config);
            break;
					case 6:
            if(dslashParam.expanding){
              sprintf(config, ",Dslash4DaggerDslash4preDaggerDslash5invDagger,partial%d,%d,%d,%d,expand%d,%d,%d,%d", 
								dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3],
                dslashParam.Rz[0], dslashParam.Rz[1], dslashParam.Rz[2], dslashParam.Rz[3]);
            }else{
              sprintf(config, ",Dslash4DaggerDslash4preDaggerDslash5invDagger,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            }
            strcat(key.aux,config);
            break;
          case 7:
            if(dslashParam.expanding){
              sprintf(config, ",Dslash4DaggerDslash4preDaggerXpay,partial%d,%d,%d,%d,expand%d,%d,%d,%d", 
								dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3],
                dslashParam.Rz[0], dslashParam.Rz[1], dslashParam.Rz[2], dslashParam.Rz[3]);
            }else{
              sprintf(config, ",Dslash4DaggerDslash4preDaggerXpay,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            }
            strcat(key.aux,config);
            break;
					case 8:
            if(dslashParam.expanding){
              sprintf(config, ",Dslash5invSm,partial%d,%d,%d,%d,expand%d,%d,%d,%d", 
								dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3],
                dslashParam.Rz[0], dslashParam.Rz[1], dslashParam.Rz[2], dslashParam.Rz[3]);
            }else{
              sprintf(config, ",Dslash5invSm,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            }
            strcat(key.aux,config);
            break;
					case 9:
            if(dslashParam.expanding){
              sprintf(config, ",Dslash5invSmTc,partial%d,%d,%d,%d,expand%d,%d,%d,%d", 
								dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3],
                dslashParam.Rz[0], dslashParam.Rz[1], dslashParam.Rz[2], dslashParam.Rz[3]);
            }else{
              sprintf(config, ",Dslash5invSmTc,partial%d,%d,%d,%d", dslashParam.R[0], dslashParam.R[1], dslashParam.R[2], dslashParam.R[3]);
            }
            strcat(key.aux,config);
            break;
        }
      
      }else{

        switch(DS_type){
          case 0:
            strcat(key.aux,",Dslash4");
            break;
          case 1:
            strcat(key.aux,",Dslash4pre");
            break;
          case 2:
            strcat(key.aux,",Dslash5");
            break;
          case 3:
            strcat(key.aux,",Dslash5inv");
            break;
          case 4:
            strcat(key.aux,",Dslash4Dslash5invDslash4pre");
            break;
					case 5:
            strcat(key.aux,",Dslash4Dslash5invXpayDslash5invDagger");
            break;
					case 6:
            strcat(key.aux,",Dslash4DaggerDslash4preDaggerDslash5invDagger");
            break;
          case 7:
            strcat(key.aux,",Dslash4DaggerDslash4preDaggerXpay");
            break;
					case 8:
            strcat(key.aux,",Dslash5invSm");
            break;
          case 9:
            strcat(key.aux,",Dslash5invSmTc");
            break;
        }
      
      }
      return key;
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      if(DS_type >= 4){ 
        // For these kernels, for one 4D-site all corresponding 5D-sites have to be within the same block,
        // since shared memory is used.
//        param.block = dim3( param.block.x, in->X(4), 1);
        param.block = dim3( 16, in->X(4), 1);
      }
      param.shared_bytes = shared_bytes_per_block(param.block.x, param.block.y);
        printfQuda( "Shared memory %08lu is larger than limit %08lu?\n", (size_t)param.shared_bytes, (size_t)(deviceProp.major>=7 ? 96*1024 : deviceProp.sharedMemPerBlock) );
//      if( (size_t)param.shared_bytes > (size_t)deviceProp.sharedMemPerBlock ) 
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      if(DS_type >= 4){ 
        // For these kernels, for one 4D-site all corresponding 5D-sites have to be within the same block,
        // since shared memory is used.
//        param.block = dim3( param.block.x, in->X(4), 1);
        param.block = dim3( 16, in->X(4), 1);
      }
      param.shared_bytes = shared_bytes_per_block(param.block.x, param.block.y);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    void apply(const hipStream_t &stream)
    {
#ifndef USE_TEXTURE_OBJECTS
      if (dslashParam.kernel_type == INTERIOR_KERNEL) bindSpinorTex<sFloat>(in, out, x);
#endif // USE_TEXTURE_OBJECTS
//      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      TuneParam tp = tuneLaunch(*this, getTuning(), QUDA_DEBUG_VERBOSE);
      setParam();
      switch(DS_type){
        case 0:
          DSLASH(MDWFDslash4, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        case 1:
          DSLASH(MDWFDslash4pre, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        case 2:
          DSLASH(MDWFDslash5, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        case 3:
          DSLASH(MDWFDslash5inv, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        case 4:
          DSLASH(MDWFDslash4Dslash5invDslash4pre, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
				case 5:
          DSLASH(MDWFDslash4Dslash5invXpayDslash5invDagger, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
				case 6:
          DSLASH(MDWFDslash4DaggerDslash4preDaggerDslash5invDagger, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        case 7:
          DSLASH(MDWFDslash4DaggerDslash4preDaggerXpay, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
				case 8:
          DSLASH(MDWFDslash5invSm, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        case 9:
          DSLASH(MDWFDslash5invSmTc, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam);
          break;
        default:
          errorQuda("invalid Dslash type");
      }
    }

    long long flops() const {
      long long Ls = in->X(4);
			long long vol4d = 0;
			if( dslashParam.partial_length ){
      	vol4d = dslashParam.partial_length;
			}else{
        vol4d = in->VolumeCB() / Ls;
			}
      long long bulk = (Ls-2)*vol4d;
      long long wall = 2*vol4d;
      long long flops = 0;
      switch(DS_type){
        case 0:
          if( dslashParam.partial_length ){
            flops = 1320ll*dslashParam.partial_length*Ls;
          }else{
            flops = DslashCuda::flops();
          }
          break;
        case 1:
          flops = 72ll*vol4d*Ls + 96ll*bulk + 120ll*wall;
          break;
        case 2:
          flops = (x ? 96ll : 48ll)*vol4d*Ls + 96ll*bulk + 120ll*wall;
          break;
        case 3:
				case 8:
				case 9:
						flops = 144ll*vol4d*Ls*Ls + 3ll*Ls*(Ls-1ll);
					break;
        case 4:
				case 6:
            flops = 1320ll*vol4d*Ls + 144ll*vol4d*Ls*Ls + 3ll*Ls*(Ls-1ll) + 72ll*vol4d*Ls + 96ll*bulk + 120ll*wall;
					break;
				case 5:
            flops = (x?1368ll:1320ll)*vol4d*Ls + 144ll*vol4d*Ls*Ls + 3ll*Ls*(Ls-1ll);
          break;
        case 7:
            flops = (x?1368ll:1320ll)*vol4d*Ls + 72ll*vol4d*Ls + 96ll*bulk + 120ll*wall;
       		break; 
				default:
          errorQuda("invalid Dslash type");
      }
      return flops;
    }

    long long bytes() const {
      bool isHalf = in->Precision() == sizeof(short) ? true : false;
      int spinor_bytes = 2 * in->Ncolor() * in->Nspin() * in->Precision() + (isHalf ? sizeof(float) : 0);
      long long Ls = in->X(4);
      long long bytes = 0;

      switch(DS_type){
        case 0:
        case 4:
				case 5:
				case 6:
        case 7:
          if( dslashParam.partial_length ){
            bytes = (x?16ll:15ll)*spinor_bytes*(long long)dslashParam.partial_length*Ls;
          }else{
            bytes = DslashCuda::bytes();
          }
          break;
        case 1:
        case 2:
          bytes = (x ? 5ll : 4ll) * spinor_bytes * in->VolumeCB();
          break;
        case 3:
				case 8:
        case 9:
          bytes = (x ? Ls + 2 : Ls + 1) * spinor_bytes * in->VolumeCB();
          break;
        default:
          errorQuda("invalid Dslash type");
      }
      return bytes;
    }
  };
#endif // GPU_DOMAIN_WALL_DIRAC

#include <dslash_policy.cuh>

  //-----------------------------------------------------
  // Modification for 4D preconditioned Mobius DWF operator
  // Additional Arg. is added to give a function name.
  //
  // pre-defined DS_type list
  // 0 = MDWF dslash4
  // 1 = MDWF dslash4pre
  // 2 = MDWF dslash5
  // 3 = MDWF dslash5inv
  //-----------------------------------------------------

  void MDWFDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge,
		      const cudaColorSpinorField *in, const int parity, const int dagger,
		      const cudaColorSpinorField *x, const double &m_f, const double &k2,
                      const double *b_5, const double *c_5, const double &m5,
		      const int *commOverride, const int DS_type, TimeProfile &profile)
  {
#ifdef GPU_DOMAIN_WALL_DIRAC
    const_cast<cudaColorSpinorField*>(in)->createComms(1);

    DslashCuda *dslash = nullptr;
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new MDWFDslashPCCuda<double2,double2>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new MDWFDslashPCCuda<float4,float4>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new MDWFDslashPCCuda<short4,short4>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    }

    // the parameters passed to dslashCuda must be 4-d volume and 3-d
    // faces because Ls is added as the y-dimension in thread space
    int ghostFace[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) ghostFace[i] = in->GhostFace()[i] / in->X(4);

    DslashPolicyImp* dslashImp = nullptr;
    if (DS_type != 0) {
      dslashImp = DslashFactory::create(QudaDslashPolicy::QUDA_DSLASH_NC);
      (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), in->Volume()/in->X(4), ghostFace, profile);
      delete dslashImp;
    } else {
      DslashPolicyTune dslash_policy(*dslash, const_cast<cudaColorSpinorField*>(in), in->Volume()/in->X(4), ghostFace, profile);
      dslash_policy.apply(0);
    }

    delete dslash;
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }
 
	void set_shared_memory_on_volta(const void* f, const char* name){
			hipDeviceProp_t device_prop;
			hipGetDeviceProperties( &device_prop, 0 );
			if(device_prop.major < 7) return;
			
			auto found = qudaFuncSetAttribute(f, hipFuncAttributeMaxDynamicSharedMemorySize, 96*1024);
			printfQuda("Found %s: %s\n", name, hipGetErrorString(found));
			
			found = qudaFuncSetAttribute(f, hipFuncAttributePreferredSharedMemoryCarveout, 100);
			printfQuda("Found %s: %s\n", name, hipGetErrorString(found));
			
			hipFuncAttributes cfa;
			found = hipFuncGetAttributes(&cfa, reinterpret_cast<const void*>(f));
			printfQuda("Found %s: %s\n", name, hipGetErrorString(found));
			
			printfQuda("Actual maximum:         %d\n", (int)cfa.maxDynamicSharedSizeBytes);
			printfQuda("Actual maximum percent: %d\n", (int)cfa.preferredShmemCarveout);
	}

  void mdwf_dslash_cuda_partial(cudaColorSpinorField *out, const cudaGaugeField &gauge,
		      const cudaColorSpinorField *in, const int parity, const int dagger,
		      const cudaColorSpinorField *x, const double &m_f, const double &k2,
                      const double *b_5, const double *c_5, const double &m5,
		      const int *commOverride, const int DS_type, TimeProfile &profile, int sp_idx_length, int R_[4], int_fastdiv Xs_[4],
          bool expanding_, std::array<int,4> Rz_)
  {
		static bool init = false;
#ifdef GPU_DOMAIN_WALL_DIRAC
    const_cast<cudaColorSpinorField*>(in)->createComms(1);

//    if(DS_type == 9){
//      hipDeviceProp_t device_prop;
//      hipGetDeviceProperties( &device_prop, 0 );
//      if(device_prop.major < 7 || in->Precision() != QUDA_HALF_PRECISION){
//        errorQuda("Your are either NOT rich enough to buy a Volta or TOO rich to buy a Volta.\n");
//      }
//    }

		if(!init){
			set_shared_memory_on_volta((const void*)MDWFDslash4Dslash5invDslash4preH18Kernel<INTERIOR_KERNEL>, 
				"MDWFDslash4Dslash5invDslash4preH18Kernel<INTERIOR_KERNEL>");
			set_shared_memory_on_volta((const void*)MDWFDslash4Dslash5invXpayDslash5invDaggerH18XpayKernel<INTERIOR_KERNEL>, 
				"MDWFDslash4Dslash5invXpayDslash5invDaggerH18XpayKernel<INTERIOR_KERNEL>");
			set_shared_memory_on_volta((const void*)MDWFDslash4DaggerDslash4preDaggerDslash5invDaggerH18Kernel<INTERIOR_KERNEL>, 
				"MDWFDslash4DaggerDslash4preDaggerDslash5invDaggerH18Kernel<INTERIOR_KERNEL>");
      set_shared_memory_on_volta((const void*)MDWFDslash5invSmTcH18DaggerKernel<INTERIOR_KERNEL>, 
				"MDWFDslash5invSmTcH18DaggerKernel<INTERIOR_KERNEL>");
			set_shared_memory_on_volta((const void*)MDWFDslash4DaggerDslash4preDaggerXpayH18XpayKernel<INTERIOR_KERNEL>, 
				"MDWFDslash4DaggerDslash4preDaggerXpayH18XpayKernel<INTERIOR_KERNEL>");
			init = true;
		}

    DslashCuda *dslash = nullptr;
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new MDWFDslashPCCuda<double2,double2>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new MDWFDslashPCCuda<float4,float4>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new MDWFDslashPCCuda<short4,short4>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    }

    dslash->dslashParam.partial_length = sp_idx_length;
    dslash->dslashParam.R[0] = R_[0];
    dslash->dslashParam.R[1] = R_[1];
    dslash->dslashParam.R[2] = R_[2];
    dslash->dslashParam.R[3] = R_[3];

    dslash->dslashParam.Xs[0] = Xs_[0];
    dslash->dslashParam.Xs[1] = Xs_[1];
    dslash->dslashParam.Xs[2] = Xs_[2];
    dslash->dslashParam.Xs[3] = Xs_[3];

//    printfQuda("volume: %dx%dx%dx%d; R: %dx%dx%dx%d; partial_length=%d.\n", 
//                                                               int(dslash->dslashParam.Xs[0]),
//                                                               int(dslash->dslashParam.Xs[1]),
//                                                               int(dslash->dslashParam.Xs[2]),
//                                                               int(dslash->dslashParam.Xs[3]), 
//                                                               int(dslash->dslashParam.R[0]), 
//                                                               int(dslash->dslashParam.R[1]), 
//                                                               int(dslash->dslashParam.R[2]), 
//                                                               int(dslash->dslashParam.R[3]), 
//                                                               sp_idx_length);
    
    if(expanding_){
      dslash->dslashParam.expanding = true;
      dslash->dslashParam.Rz[0] = Rz_[0];
      dslash->dslashParam.Rz[1] = Rz_[1];
      dslash->dslashParam.Rz[2] = Rz_[2];
      dslash->dslashParam.Rz[3] = Rz_[3];
    } 

    // the parameters passed to dslashCuda must be 4-d volume and 3-d
    // faces because Ls is added as the y-dimension in thread space
    int ghostFace[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) ghostFace[i] = in->GhostFace()[i] / in->X(4);

    DslashPolicyImp* dslashImp = nullptr;
    if (DS_type != 0) {
      dslashImp = DslashFactory::create(QudaDslashPolicy::QUDA_DSLASH_NC);
      (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), sp_idx_length, ghostFace, profile);
      delete dslashImp;
    } else {
      DslashPolicyTune dslash_policy(*dslash, const_cast<cudaColorSpinorField*>(in), sp_idx_length, ghostFace, profile);
      dslash_policy.apply(0);
    }
    // sp_idx_length is the param.threads

    delete dslash;
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }
  
  void mdwf_dslash_cuda_partial_tc(cudaColorSpinorField *out, const cudaGaugeField &gauge,
		      const cudaColorSpinorField *in, const int parity, const int dagger,
		      const cudaColorSpinorField *x, const double &m_f, const double &k2,
                      const double *b_5, const double *c_5, const double &m5,
		      const int *commOverride, const int DS_type, TimeProfile &profile, int sp_idx_length, int R_[4], int_fastdiv Xs_[4],
          void* gpu_m5inv, int_fastdiv Ms_)
  {
		static bool init = false;
#ifdef GPU_DOMAIN_WALL_DIRAC
    const_cast<cudaColorSpinorField*>(in)->createComms(1);

//    if(DS_type == 9){
//      hipDeviceProp_t device_prop;
//      hipGetDeviceProperties( &device_prop, 0 );
//      if(device_prop.major < 7 || in->Precision() != QUDA_HALF_PRECISION){
//        errorQuda("Your are either NOT rich enough to buy a Volta or TOO rich to buy a Volta.\n");
//      }
//    }

		if(!init){
			set_shared_memory_on_volta((const void*)MDWFDslash4Dslash5invDslash4preH18Kernel<INTERIOR_KERNEL>, 
				"MDWFDslash4Dslash5invDslash4preH18Kernel<INTERIOR_KERNEL>");
			set_shared_memory_on_volta((const void*)MDWFDslash4Dslash5invXpayDslash5invDaggerH18XpayKernel<INTERIOR_KERNEL>, 
				"MDWFDslash4Dslash5invXpayDslash5invDaggerH18XpayKernel<INTERIOR_KERNEL>");
			set_shared_memory_on_volta((const void*)MDWFDslash4DaggerDslash4preDaggerDslash5invDaggerH18Kernel<INTERIOR_KERNEL>, 
				"MDWFDslash4DaggerDslash4preDaggerDslash5invDaggerH18Kernel<INTERIOR_KERNEL>");
      set_shared_memory_on_volta((const void*)MDWFDslash5invSmTcH18DaggerKernel<INTERIOR_KERNEL>, 
				"MDWFDslash5invSmTcH18DaggerKernel<INTERIOR_KERNEL>");
			set_shared_memory_on_volta((const void*)MDWFDslash4DaggerDslash4preDaggerXpayH18XpayKernel<INTERIOR_KERNEL>, 
				"MDWFDslash4DaggerDslash4preDaggerXpayH18XpayKernel<INTERIOR_KERNEL>");
			init = true;
		}

    DslashCuda *dslash = nullptr;
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
      dslash = new MDWFDslashPCCuda<double2,double2>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new MDWFDslashPCCuda<float4,float4>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new MDWFDslashPCCuda<short4,short4>(out, gauge, in, x, m_f, k2, b_5, c_5, m5, parity, dagger, commOverride, DS_type);
    }

    dslash->dslashParam.m5inv = gpu_m5inv;
    dslash->dslashParam.Ms = Ms_;

    dslash->dslashParam.partial_length = sp_idx_length;
    dslash->dslashParam.R[0] = R_[0];
    dslash->dslashParam.R[1] = R_[1];
    dslash->dslashParam.R[2] = R_[2];
    dslash->dslashParam.R[3] = R_[3];

    dslash->dslashParam.Xs[0] = Xs_[0];
    dslash->dslashParam.Xs[1] = Xs_[1];
    dslash->dslashParam.Xs[2] = Xs_[2];
    dslash->dslashParam.Xs[3] = Xs_[3];

//    printfQuda("volume: %dx%dx%dx%d; R: %dx%dx%dx%d; partial_length=%d.\n", 
//                                                               int(dslash->dslashParam.Xs[0]),
//                                                               int(dslash->dslashParam.Xs[1]),
//                                                               int(dslash->dslashParam.Xs[2]),
//                                                               int(dslash->dslashParam.Xs[3]), 
//                                                               int(dslash->dslashParam.R[0]), 
//                                                               int(dslash->dslashParam.R[1]), 
//                                                               int(dslash->dslashParam.R[2]), 
//                                                               int(dslash->dslashParam.R[3]), 
//                                                               sp_idx_length);
    
    // the parameters passed to dslashCuda must be 4-d volume and 3-d
    // faces because Ls is added as the y-dimension in thread space
    int ghostFace[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) ghostFace[i] = in->GhostFace()[i] / in->X(4);

    DslashPolicyImp* dslashImp = nullptr;
    if (DS_type != 0) {
      dslashImp = DslashFactory::create(QudaDslashPolicy::QUDA_DSLASH_NC);
      (*dslashImp)(*dslash, const_cast<cudaColorSpinorField*>(in), sp_idx_length, ghostFace, profile);
      delete dslashImp;
    } else {
      DslashPolicyTune dslash_policy(*dslash, const_cast<cudaColorSpinorField*>(in), sp_idx_length, ghostFace, profile);
      dslash_policy.apply(0);
    }
    // sp_idx_length is the param.threads

    delete dslash;
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }

}
